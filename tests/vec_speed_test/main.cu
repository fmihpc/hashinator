#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <vector>
#include <chrono>
#include "../../src/hashinator/hashinator.h"
#include "/home/kostis/dev/profiny/Profiny.h"

#define N 1
#define elems 1<<29

typedef split::SplitVector<int> splitvector ;
typedef std::vector<int> stdvector ;


__global__ 
void stress_kernel(splitvector*a, splitvector* b, splitvector* c){
   
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index< a->size()){
      c->at(index)=a->at(index)+b->at(index);
   }
}


__global__
void change_if(splitvector*a){
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i< a->size()){
      if ( a->operator[](i) >1 ){
         a->at(i)=0;
      }
   }
}

void gpu_stress_test(){
	PROFINY_SCOPE

   splitvector a(elems,1);
   splitvector b(elems,2);
   splitvector c(elems,0);

   splitvector* d_a=a.upload();
   splitvector* d_b=b.upload();
   splitvector* d_c=c.upload();

   stress_kernel<<<elems,32>>>(d_a,d_b,d_c);
   hipDeviceSynchronize();
   change_if<<<elems,32>>>(d_a);
   hipDeviceSynchronize();
   


   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);

}

void stress_test_std(stdvector& vec,const size_t elements){
	PROFINY_SCOPE
   for (int ntimes=0; ntimes<N; ntimes++){
      //Load with pushbacks
      for (size_t i=0; i<elements;i++){
         vec.push_back(i);
      }
      //Read random
      const size_t size=vec.size();
      for (size_t i=0; i<elements;i++){
         vec[i]+=1;
         vec[size-i]+=1;
      }
      //Pop all elements
      for (size_t i=0; i<elements;i++){
         vec.pop_back();
      }
   }
   vec=stdvector();
}

void stress_test_split(splitvector& vec,const size_t elements){
	PROFINY_SCOPE
   for (int ntimes=0; ntimes<N; ntimes++){
      //Load with pushbacks
      for (size_t i=0; i<elements;i++){
         vec.push_back(i);
      }
      //Read random
      const size_t size=vec.size();
      for (size_t i=0; i<elements;i++){
         vec[i]+=1;
         vec[size-i]+=1;
      }
      //Pop all elements
      for (size_t i=0; i<elements;i++){
         vec.pop_back();
      }
   }
   vec=splitvector();
}


int main(int argc, char** argv){

   splitvector a;
   stdvector b;

   
	PROFINY_SCOPE
	profiny::Profiler::setOmitRecursiveCalls(false);
   stress_test_split(a,1e9);
   stress_test_std(b,1e9);

   gpu_stress_test();

}
