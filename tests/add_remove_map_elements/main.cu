#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>
#include "../../src/hashinator/hashinator.h"
#include <gtest/gtest.h>
#define N 1<<12

typedef uint32_t val_type;


void cpu_write_map(Hashinator<val_type,val_type>& map,int total_keys=N){
   for (val_type i=0; i<total_keys;i++){
      map[i]=0;
   }
}

__global__
void gpu_write_map(Hashinator<val_type,val_type> *dmap){
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index<N){
      dmap->set_element(index,index);
   }
   return;
}


__global__
void gpu_delete_all(Hashinator<val_type,val_type> *dmap){
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index<N ){
      auto kpos=dmap->d_find(index);
      dmap->d_erase(kpos);
   }
   return;
}


__global__
void gpu_delete_even(Hashinator<val_type,val_type> *dmap){
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index<N ){
      auto kpos=dmap->d_find(index);
      if (kpos==dmap->d_end()){return;}
      if (kpos->second %2==0 ){
         dmap->d_erase(kpos);
      }
   }
   return;
}


__global__
void gpu_delete_odd(Hashinator<val_type,val_type> *dmap){
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index<N ){
      auto kpos=dmap->d_find(index);
      if (kpos==dmap->d_end()){return;}
      if (kpos->second %2==1){
         dmap->d_erase(kpos);
      }
   }
   return;
}


int main(int argc, char* argv[]){

   //We create an instance of hashinator and add elements to it on host
   Hashinator<val_type,val_type> hmap;
   cpu_write_map(hmap,N);

   //Some magic numbers!( used to launch the kernels)
   size_t threads=32;
   size_t blocks=1<<10;

   //Declare a pointer for use in kernels
   Hashinator<val_type,val_type>* dmap;

   //Upload map to device
   dmap=hmap.upload();

   //Call a simple kernel that just writes to the map elements based on their index
   gpu_write_map<<<blocks,threads>>> (dmap);
   hipDeviceSynchronize();
   
   //Always clean up after kernel
   hmap.clean_up_after_device(dmap);

   //Let's reupload the map
   dmap=hmap.upload();

   //Now we delete all even elements
   gpu_delete_even<<<blocks,threads>>> (dmap);
   hipDeviceSynchronize();

   //And we clean up again
   hmap.clean_up_after_device(dmap);

   //One more time
   dmap=hmap.upload();
   //And we remove the odd numbers
   gpu_delete_odd<<<blocks,threads>>> (dmap);
   hipDeviceSynchronize();

   //We clean up
   hmap.clean_up_after_device(dmap);

   //We now expect the map to have 0 fill as we deleted all the elemets
   std::cout<<"Map should have 0 fill:\n";
   std::cout<<"Map's fill is -->"<<hmap.size()<<std::endl;
   assert(hmap.size()==0 && "Map should have zero 0 but that is not the case!");




   return 0;
}

