#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>
#include <gtest/gtest.h>
#include "../../src/splitvector/splitvec.h"
#include <hip/hip_runtime_api.h>

#define expect_true EXPECT_TRUE
#define expect_false EXPECT_FALSE
#define expect_eq EXPECT_EQ
#define N 1<<12

typedef split::SplitVector<int> vec ;
typedef split::SplitVector<split::SplitVector<int>> vec2d ;

__global__
void add_vectors(vec* a , vec* b,vec* c){

   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index< a->size()){
      c->at(index)=a->at(index)+b->at(index);
   }

}

__global__
void push_back_kernel(vec* a){

   int index = blockIdx.x * blockDim.x + threadIdx.x;
   a->push_back(index);

}

TEST(Test_Dealloc_Pattern, Ctor_Dtor){

   vec a;
   a.push_back(1);
   a.clear();
   a.reserve(32);
   printf("Before kernel size = %zu , capacity = %zu \n",a.size(),a.capacity());
   vec* d_a=a.upload();
   push_back_kernel<<<4,8>>>(d_a);
   hipDeviceSynchronize();
   hipFree(d_a);
   printf("After kernel size = %zu , capacity = %zu \n",a.size(),a.capacity());

}


TEST(Test_GPU,VectorAddition){
   vec a(N,1);
   vec b(N,2);
   vec c(N,0);
   
   vec* d_a=a.upload();
   vec* d_b=b.upload();
   vec* d_c=c.upload();

   add_vectors<<<N,32>>>(d_a,d_b,d_c);
   hipDeviceSynchronize();
   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);


   for (const auto& e:c){
      expect_true(e==3);
   }


}

TEST(Test_2D_Contruct,VecOfVec){

   vec inner_a(100,1);
   vec inner_b(100,2);
   vec2d a(10,inner_a);
   vec2d b(10,inner_b);

   for (auto &i:a){
      for (const auto &val:i){
         EXPECT_EQ(val, 1);
      }
   }
   for (auto &i:b){
      for (const auto &val:i){
         EXPECT_EQ(val, 2);
      }
   }
   expect_true(a!=b);
   expect_true(a!=b);
   expect_false(a==b);
   expect_false(a==b);
}

TEST(Constructors,Default){
   vec a;
   expect_true(a.size()==0 && a.capacity()==0);
   expect_true(a.data()==nullptr);
}

TEST(Constructors,Size_based){
   vec a(N);
   expect_true(a.size()==N && a.capacity()==N);
   expect_true(a.data()!=nullptr);
}

TEST(Constructors,std_vector){
   std::vector<int>  stdvec(N,10);
   vec a(stdvec);

   for (size_t i=0; i<N; i++){
      expect_true(stdvec[i]=a[i]);
   }
   vec b(a);
   expect_true(a==b);
}

TEST(Constructors,Specific_Value){
   vec a(N,5);
   expect_true(a.size()==N && a.capacity()==N);
   for (size_t i=0; i<N;i++){
      expect_true(a[i]==5);
      expect_true(a.at(i)==5);
   }
}

TEST(Constructors,Copy){
   vec a(N,5);
   vec b(a);
   for (size_t i=0; i<N;i++){
      expect_true(a[i]==b[i]);
      expect_true(a.at(i)==b.at(i));
   }
}

TEST(Vector_Functionality , Reserve){
   vec a;
   size_t cap =1000000;
   a.reserve(cap);
   expect_true(a.size()==0);
   expect_true(a.capacity()==cap);
}

TEST(Vector_Functionality , Resize){
   vec a;
   size_t size =1<<20;
   a.resize(size);
   expect_true(a.size()==size);
   expect_true(a.capacity()==a.size());
}

TEST(Vector_Functionality , Swap){
   vec a(10,2),b(10,2);
   a.swap(b);
   expect_true(a==b);
   vec c(100,1);
   vec d (200,3);
   c.swap(d);
   expect_true(c.size()==200);
   expect_true(d.size()==100);
   expect_true(c.front()==3);
   expect_true(d.front()==1);

}

TEST(Vector_Functionality , Resize2){
   vec a;
   size_t size =1<<20;
   a.resize(size);
   expect_true(a.size()==size);
   expect_true(a.capacity()==a.size());
}

TEST(Vector_Functionality , Clear){
   vec a(10);
   size_t size =1<<20;
   a.resize(size);
   expect_true(a.size()==size);
   auto cap=a.capacity();
   a.clear();
   expect_true(a.size()==0);
   expect_true(a.capacity()==cap);
}

TEST(Vector_Functionality , PopBack){
   vec a{1,2,3,4,5,6,7,8,9,10};
   size_t initial_size=a.size();
   size_t initial_cap=a.capacity();
   for (int i=9;i>=0;i--){
      a.pop_back();
      if (a.size()>0){
         expect_true(i==a.back());
      }
   }
   expect_true(a.size()==0);
   expect_true(a.capacity()==initial_cap);
}

TEST(Vector_Functionality , Push_Back){
   vec a;
   for (auto i=a.begin(); i!=a.end();i++){
      expect_true(false);
   }

   size_t initial_size=a.size();
   size_t initial_cap=a.capacity();

   a.push_back(11);
   expect_true(11==a[a.size()-1]);
   a.push_back(12);
   expect_true(12==a[a.size()-1]);

}


TEST(Vector_Functionality , Shrink_to_Fit){
   vec a;
   for (auto i=a.begin(); i!=a.end();i++){
      expect_true(false);
   }

   size_t initial_size=a.size();
   size_t initial_cap=a.capacity();

   for (int i =0 ; i< 1024; i++){
      a.push_back(i);
   }

   expect_true(a.size()<a.capacity());
   a.shrink_to_fit();
   expect_true(a.size()==a.capacity());

}
TEST(Vector_Functionality , Push_Back_2){
   vec a{1,2,3,4,5,6,7,8,9,10};
   size_t initial_size=a.size();
   size_t initial_cap=a.capacity();

   a.push_back(11);
   expect_true(11==a[a.size()-1]);
   a.push_back(12);
   expect_true(12==a[a.size()-1]);

}

TEST(Vector_Functionality , Insert_0){
   {
      split::SplitVector<int> a{1,2,3,4,5,6,7,8,9,10};
      auto s0=a.size(); auto c0=a.capacity();
      auto it(a.begin());
      auto it2=a.insert(it,-1);
      expect_true(a[0]=-1);
      expect_true(a.size()==s0+1);
      expect_true(a.capacity()>c0);
   }
   {
      vec a{1,2,3,4,5,6,7,8,9,10};
      auto s0=a.size(); auto c0=a.capacity();
      vec::iterator it(a.end());
      auto it2=a.insert(it,-1);
      expect_true(a.back()=-1);
      expect_true(a[a.size()-1]=-1);
      expect_true(a.size()==s0+1);
      expect_true(a.capacity()>c0);
   }
   {
      vec a{1,2,3,4,5,6,7,8,9,10};
      auto s0=a.size(); auto c0=a.capacity();
      vec::iterator it(&a[4]);
      auto it2=a.insert(it,-1);
      expect_true(a[4]=-1);
      expect_true(a.size()==s0+1);
      expect_true(a.capacity()>c0);
   }
   {
     vec a{1,2,3,4,5,6,7,8,9,10};
     auto s0=a.size(); auto c0=a.capacity();
     try {
      //hehe
      vec::iterator it(nullptr);
      auto it2=a.insert(it,-1);
     }// this has to throw
     catch (...) {
        expect_true(true);
        expect_true(a.capacity()==c0);
        expect_true(a.size()==s0);
           return;
     }
     //if we end up here it never threw so something's up
     expect_true(false);
   }
}






int main(int argc, char* argv[]){
   ::testing::InitGoogleTest(&argc, argv);
   return RUN_ALL_TESTS();
}
