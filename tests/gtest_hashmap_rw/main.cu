#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>
#include "../../src/hashinator/hashinator.h"
#include <gtest/gtest.h>
#define N  64

typedef uint32_t val_type;


__global__
void gpu_read_map(Hashinator<val_type,val_type> *dmap){
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   const val_type& element=dmap->read_element(index);
   assert(element == index);
}


__global__
void gpu_write_map(Hashinator<val_type,val_type> *dmap){
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   dmap->set_element(index,index);

}

__global__
void addtoMap(Hashinator<val_type,val_type> *dmap){
   int index = (blockIdx.x * blockDim.x + threadIdx.x)+268435456;
    dmap->set_element(index, index);
}

void cpu_write_map(Hashinator<val_type,val_type>& map,int total_keys=N){
   for (val_type i=0; i<total_keys;i++){
      map[i]=i;
   }
}

void addNelems(Hashinator<val_type,val_type>&map,int numel){
   int threads=32;
   auto initial_lf=map.load_factor();
   int blocks=numel/threads;
   //printf("Adding %i elements  with %i Threads and %i Blocks\n ",(int)numel,(int)threads,(int)blocks);
   Hashinator<val_type,val_type>* dmap = map.upload();
   auto start = std::chrono::high_resolution_clock::now();
   addtoMap<<<blocks,threads>>>(dmap);
   hipDeviceSynchronize();
   auto end = std::chrono::high_resolution_clock::now();
   map.clean_up_after_device(dmap);
   //map.print_all();
   auto total_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
   printf("--->TIME: %.3f seconds for %zu elements at a load factor of %f\n", total_time.count() * 1e-9,map.size(),initial_lf);
}
   
   


TEST(CPU_TEST,CPU_Read_Check){
   Hashinator<val_type,val_type> map;
   cpu_write_map(map);
   for (auto &kval:map){
      EXPECT_TRUE(kval.first == kval.second);
   }
}   

TEST(GPU_TEST,GPU_Read_Check){
   Hashinator<val_type,val_type> map;
   cpu_write_map(map);
   for (auto &kval:map){
      EXPECT_TRUE(kval.first == kval.second);
   }

   Hashinator<val_type,val_type>* dmap = map.upload();
   gpu_read_map<<<4,4>>>(dmap);
   hipDeviceSynchronize();
   map.clean_up_after_device(dmap);
}   


TEST(GPU_TEST,GPU_Write_Check){
   Hashinator<val_type,val_type> map;
   cpu_write_map(map,1<<20);
   for (auto &kval:map){
      EXPECT_TRUE(kval.first == kval.second);
   }
   map.print_all();
   addNelems(map, 1000000);
   
   for (auto &kval:map){
      EXPECT_TRUE(kval.first == kval.second);
   }
   
   Hashinator<val_type,val_type>* dmap = map.upload();
   gpu_read_map<<<4,4>>>(dmap);
   hipDeviceSynchronize();
   map.clean_up_after_device(dmap);
   map.print_all();

}   


TEST(GPU_TEST,GPU_Read_Write_Check){
   Hashinator<val_type,val_type> map;
   map.resize(29);
   
   Hashinator<val_type,val_type>* dmap = map.upload();
   size_t blocks=(1<<7)/32;
   std::cout<<"DONE"<<std::endl;
   std::cout<<blocks<<std::endl;

   gpu_write_map<<<blocks,32>>>(dmap);
   hipDeviceSynchronize();
   gpu_read_map<<<blocks,32>>>(dmap);
   hipDeviceSynchronize();
   map.clean_up_after_device(dmap);
   map.print_all();

}   

int main(int argc, char* argv[]){
   ::testing::InitGoogleTest(&argc, argv);
   return RUN_ALL_TESTS();
}

