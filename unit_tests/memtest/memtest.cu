#include "hip/hip_runtime.h"
#include "memtest.h"

/* Memory reporting function
 */
void gpu_reportMemory(const std::vector<testStructure> &objs) {
   size_t o_size=0;
   size_t o_cap=0;
   const size_t nob=objs.size();
   std::cerr<<" nob "<<nob<<std::endl;
   for (uint i=0; i<nob; ++i) {
      o_size += (objs.at(i)).size();
      o_cap += (objs.at(i)).capacity();
   }
   
   size_t free_byte ;
   size_t total_byte ;
   SPLIT_CHECK_ERR( hipMemGetInfo( &free_byte, &total_byte) );
   size_t used_mb = (total_byte-free_byte)/(1024*1024);
   std::cerr<<" =================================="<<std::endl;
   std::cerr<<" GPU Memory report"<<std::endl;
   std::cerr<<"   objects size:            "<<o_size/(1024*1024)<<" Mbytes"<<std::endl;
   std::cerr<<"   objects capacity:        "<<o_cap/(1024*1024)<<" Mbytes"<<std::endl;
   std::cerr<<"   Reported Hardware use:   "<<used_mb<<" Mbytes"<<std::endl;
   std::cerr<<" =================================="<<std::endl;
   return;
}

TEST(Test_GPU,Memory) {
   //int myDevice;
   const int n_objs = 20;
   const int n_loops = 27; 

   std::vector<testStructure> storage;

   std::cerr<<"pre-init"<<std::endl;
   gpu_reportMemory(storage);
   
   //SPLIT_CHECK_ERR( gpuGetDevice(&myDevice) );
   for (uint i=0; i<n_objs; ++i) {
      std::cerr<<"init "<<i<<std::endl;
      storage.push_back(testStructure());
   }

   std::cerr<<"initial"<<std::endl;
   const int initial_size = storage[0].size();
   gpu_reportMemory(storage);

   //final_size = std::pow(2,35) // 34 gigs
   for (uint j=0; j<n_loops; ++j) {
      size_t newSize = initial_size * std::pow(2,j);

      std::cerr<<" recapacitate "<<j<<std::endl;
      for (uint i=0; i<n_objs; ++i) {
         storage[i].recapacitate(newSize);
      }
      gpu_reportMemory(storage);
      
      std::cerr<<" resize "<<j<<std::endl;
      for (uint i=0; i<n_objs; ++i) {
         storage[i].resize(newSize-1);
      }
      gpu_reportMemory(storage);
   }

   SPLIT_CHECK_ERR( split_gpuDeviceSynchronize() );
   EXPECT_TRUE(true);
}
__host__ int main(int argc, char* argv[]) {
   ::testing::InitGoogleTest(&argc, argv);
   return RUN_ALL_TESTS();
}

