#include "hip/hip_runtime.h"
#include "memtest.h"

size_t base_usage = 0;
size_t mem_limit = 6 * std::pow(1024,3);

/* Memory reporting function
 */
void gpu_reportMemory(const std::vector<testStructure> &objs) {
   size_t o_size=0;
   size_t o_cap=0;
   const size_t nob=objs.size();
   for (uint i=0; i<nob; ++i) {
      o_size += (objs.at(i)).sizeInBytes();
      o_cap += (objs.at(i)).capacityInBytes();
   }
   
   size_t free_byte ;
   size_t total_byte ;
   SPLIT_CHECK_ERR( hipMemGetInfo( &free_byte, &total_byte) );
   size_t used_mb = (total_byte-free_byte)/(1024*1024);

   // store base usage at start
   if (nob==0) {
//      base_usage = used_mb;
      std::cerr<<" =================================="<<std::endl;
      std::cerr<<" GPU Memory: base usage is "<<base_usage<<" Mbytes"<<std::endl;
      std::cerr<<" =================================="<<std::endl;
      return;
   }
   int64_t int_used_mb = (int64_t)used_mb - (int64_t)base_usage;    
   std::cerr<<" =================================="<<std::endl;
   std::cerr<<" GPU Memory report"<<std::endl;
   std::cerr<<"   objects size:            "<<o_size/(1024*1024)<<" Mbytes"<<std::endl;
   std::cerr<<"   objects capacity:        "<<o_cap/(1024*1024)<<" Mbytes"<<std::endl;
   std::cerr<<"   Reported Hardware use:   "<<int_used_mb<<" Mbytes"<<std::endl;
   std::cerr<<" =================================="<<std::endl;
   return;
}

TEST(Test_GPU,Memory) {
   //int myDevice;
   const int n_objs = 20;
   const int n_loops = 10;

   std::vector<testStructure> storage;

   gpu_reportMemory(storage);
   
   //SPLIT_CHECK_ERR( gpuGetDevice(&myDevice) );
   for (uint i=0; i<n_objs; ++i) {
      storage.push_back(testStructure(1024*1024));
   }

   const int initial_size = storage[0].size();
   gpu_reportMemory(storage);

   //final_size = std::pow(2,35) // 34 gigs
   for (uint j=0; j<n_loops; ++j) {
      size_t newSize = initial_size * std::pow(2,j);

      if (n_objs * newSize > mem_limit) {
         break;
      }   
      std::cerr<<"============  CYCLE "<<j<<" ==============="<<std::endl;


      std::cerr<<" recapacitate "<<j<<std::endl;
      for (uint i=0; i<n_objs; ++i) {
         storage[i].recapacitate(newSize);
      }
      gpu_reportMemory(storage);
      
      std::cerr<<" resize "<<j<<std::endl;
      for (uint i=0; i<n_objs; ++i) {
         storage[i].resize(newSize-1);
      }
      gpu_reportMemory(storage);
      
      std::cerr<<" resize down "<<j<<std::endl;
      for (uint i=0; i<n_objs; ++i) {
         storage[i].resize(newSize/1024);
      }
      gpu_reportMemory(storage);

      std::cerr<<" shrink_to_fit 1 "<<j<<std::endl;
      for (uint i=0; i<n_objs; ++i) {
         storage[i].shrink_to_fit();
      }
      gpu_reportMemory(storage);

      std::cerr<<std::endl<<std::endl;

      std::cerr<<" resize again "<<j<<std::endl;
      for (uint i=0; i<n_objs; ++i) {
         storage[i].resize(newSize-1);
      }
      gpu_reportMemory(storage);      
      std::cerr<<" resize down again "<<j<<std::endl;
      for (uint i=0; i<n_objs; ++i) {
         storage[i].resize(newSize/1024);
      }
      gpu_reportMemory(storage);
      std::cerr<<" shrink_to_fit 2 "<<j<<std::endl;
      for (uint i=0; i<n_objs; ++i) {
         storage[i].shrink_to_fit_2();
      }
      gpu_reportMemory(storage);
}

   SPLIT_CHECK_ERR( split_gpuDeviceSynchronize() );
   EXPECT_TRUE(true);
}
__host__ int main(int argc, char* argv[]) {
   ::testing::InitGoogleTest(&argc, argv);
   return RUN_ALL_TESTS();
}

