#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>
#include <gtest/gtest.h>
#include "../../include/splitvector/splitvec.h"
#include "../../include/splitvector/split_tools.h"
#include "umpire/Allocator.hpp"
#include "umpire/ResourceManager.hpp"
#include "umpire/TypedAllocator.hpp"


#define expect_true EXPECT_TRUE
#define expect_false EXPECT_FALSE
#define expect_eq EXPECT_EQ
#define N 1<<12

static umpire::TypedAllocator<int>* vector_alloc;
typedef split::SplitVector<int,umpire::TypedAllocator<int>> vec ;
// typedef split::SplitVector<int,split::split_unified_allocator<int>> vec ;


__global__
void add_vectors(vec* a , vec* b,vec* c){

   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index< a->size()){
      c->at(index)=a->at(index)+b->at(index);
   }

}


__global__
void resize_vector(vec* a , int size){
   a->device_resize(size);
}


__global__
void push_back_kernel(vec* a){

   int index = blockIdx.x * blockDim.x + threadIdx.x;
   a->device_push_back(index);
}

__global__
void merge_kernel(vec* a,vec *b ){

   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index==0){
      a->device_insert(a->end(),b->begin(),b->end());
   }
}

__global__
void merge_kernel_2(vec* a){

   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index==0){
      a->device_insert(a->begin()++,3,42);
   }
}

__global__
void erase_kernel(vec* a){
   auto it=a->begin();
   a->erase(it);
   
}



void print_vec_elements(const vec& v){
   std::cout<<"****Vector Contents********"<<std::endl;
   std::cout<<"Size= "<<v.size()<<std::endl;
   std::cout<<"Capacity= "<<v.capacity()<<std::endl;
   for (const auto i:v){
      std::cout<<i<<" ";
   }

   std::cout<<"\n****~Vector Contents********"<<std::endl;
}

TEST(Test_GPU,VectorAddition){
   vec a(N,1,*vector_alloc);
   vec b(N,2,*vector_alloc);
   vec c(N,0,*vector_alloc);
   
   vec* d_a=a.upload();
   vec* d_b=b.upload();
   vec* d_c=c.upload();

   add_vectors<<<N,32>>>(d_a,d_b,d_c);
   SPLIT_CHECK_ERR( split_gpuDeviceSynchronize() );

   for (const auto& e:c){
      expect_true(e==3);
   }


}

TEST(Constructors,Default){
   vec a(*vector_alloc);
   expect_true(a.size()==0 && a.capacity()==0);
   expect_true(a.data()==nullptr);
}

TEST(Constructors,Size_based){
   vec a(N,*vector_alloc);
   expect_true(a.size()==N && a.capacity()==N);
   expect_true(a.data()!=nullptr);
}


TEST(Constructors,Specific_Value){
   vec a(N,5,*vector_alloc);
   expect_true(a.size()==N && a.capacity()==N);
   for (size_t i=0; i<N;i++){
      expect_true(a[i]==5);
      expect_true(a.at(i)==5);
   }
}

TEST(Constructors,Copy){
   vec a(N,5,*vector_alloc);
   vec b(a);
   for (size_t i=0; i<N;i++){
      expect_true(a[i]==b[i]);
      expect_true(a.at(i)==b.at(i));
   }
}

TEST(Vector_Functionality , Reserve){
   vec a(*vector_alloc);
   size_t cap =1000000;
   a.reserve(cap);
   expect_true(a.size()==0);
   expect_true(a.capacity()==cap);
}

TEST(Vector_Functionality , Resize){
   vec a(*vector_alloc);
   size_t size =1<<20;
   a.resize(size);
   expect_true(a.size()==size);
   expect_true(a.capacity()==a.size());
}

TEST(Vector_Functionality , Swap){
   vec a(10,2,*vector_alloc);
   vec b(10,2,*vector_alloc);
   a.swap(b);
   vec c(100,1,*vector_alloc);
   vec d (200,3,*vector_alloc);
   c.swap(d);
   expect_true(c.size()==200);
   expect_true(d.size()==100);
   expect_true(c.front()==3);
   expect_true(d.front()==1);

}

TEST(Vector_Functionality , Resize2){
   vec a(*vector_alloc);
   size_t size =1<<20;
   a.resize(size);
   expect_true(a.size()==size);
   expect_true(a.capacity()==a.size());
}

TEST(Vector_Functionality , Clear){
   vec a(10,*vector_alloc);
   size_t size =1<<20;
   a.resize(size);
   expect_true(a.size()==size);
   auto cap=a.capacity();
   a.clear();
   expect_true(a.size()==0);
   expect_true(a.capacity()==cap);
}


TEST(Vector_Functionality , Push_Back){
   vec a(*vector_alloc);
   for (auto i=a.begin(); i!=a.end();i++){
      expect_true(false);
   }

   size_t initial_size=a.size();
   size_t initial_cap=a.capacity();

   a.push_back(11);
   expect_true(11==a[a.size()-1]);
   a.push_back(12);
   expect_true(12==a[a.size()-1]);

}


TEST(Vector_Functionality , Shrink_to_Fit){
   vec a(*vector_alloc);
   for (auto i=a.begin(); i!=a.end();i++){
      expect_true(false);
   }

   size_t initial_size=a.size();
   size_t initial_cap=a.capacity();

   for (int i =0 ; i< 1024; i++){
      a.push_back(i);
   }

   expect_true(a.size()<a.capacity());
   a.shrink_to_fit();
   expect_true(a.size()==a.capacity());

}

TEST(Vector_Functionality , PushBack_And_Erase_Device){
      vec a(*vector_alloc);
      a.reserve(100);
      vec* d_a=a.upload();
      push_back_kernel<<<4,8>>>(d_a);
      SPLIT_CHECK_ERR( split_gpuDeviceSynchronize() );
      vec* d_b=a.upload();
      erase_kernel<<<1,1>>>(d_b);
      SPLIT_CHECK_ERR( split_gpuDeviceSynchronize() );
}



TEST(Vector_Functionality , Resizing_Device){

   {
      vec a(32,42,*vector_alloc);
      expect_true(a.size()==a.capacity());
      a.resize(16);
      expect_true(a.size()==16);
      expect_true(a.capacity()==32);
   }

   {
      vec a(32,42,*vector_alloc);
      expect_true(a.size()==a.capacity());
      vec* d_a=a.upload();
      resize_vector<<<1,1>>>(d_a,16);
      SPLIT_CHECK_ERR( split_gpuDeviceSynchronize() );
      expect_true(a.size()==16);
      expect_true(a.capacity()==32);
   }


   {
      vec a(32,42,*vector_alloc);
      expect_true(a.size()==a.capacity());
      a.reserve(100);
      expect_true(a.capacity()>100);
      vec* d_a=a.upload();
      resize_vector<<<1,1>>>(d_a,64);
      SPLIT_CHECK_ERR( split_gpuDeviceSynchronize() );
      expect_true(a.size()==64);
      expect_true(a.capacity()>100);
      for (size_t i = 0 ; i< a.size(); ++i){
         a.at(i)=3;
         expect_true(a.at(i)=3);
      }
   }
}

TEST(Vector_Functionality , Test_CopyMetaData){

   vec a(32,42,*vector_alloc);
   expect_true(a.size()==a.capacity());
   a.resize(16);
   expect_true(a.size()==16);
   expect_true(a.capacity()==32);
   split::SplitInfo* info;
   SPLIT_CHECK_ERR( split_gpuMallocHost((void **) &info, sizeof(split::SplitInfo)) );
   a.copyMetadata(info);
   SPLIT_CHECK_ERR( split_gpuDeviceSynchronize() );
   expect_true(a.capacity()==info->capacity);
   expect_true(a.size()==info->size);
}


int main(int argc, char* argv[]){
   auto& rm = umpire::ResourceManager::getInstance();
   umpire::Allocator alloc = rm.getAllocator("UM");
   auto va=umpire::TypedAllocator<int>(alloc);   
   vector_alloc=&va;
   ::testing::InitGoogleTest(&argc, argv);
   return RUN_ALL_TESTS();
}
