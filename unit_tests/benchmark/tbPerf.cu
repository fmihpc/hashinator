#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>
#include <unordered_set>
#include <random>
#include "../../include/hashinator/hashinator.h"
constexpr int R = 10;

using namespace std::chrono;
using namespace Hashinator;
typedef uint32_t val_type;
typedef uint32_t key_type;
typedef split::SplitVector<hash_pair<key_type,val_type>> vector ;
typedef split::SplitVector<key_type> key_vec;
typedef split::SplitVector<val_type> val_vec;
using hashmap= Hashmap<key_type,val_type>;



auto generateNonDuplicatePairs(vector& src,const size_t size)->void {
    std::unordered_set<int> keys;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<val_type> dist(1, std::numeric_limits<val_type>::max());

    src.clear();
    while (src.size() < size) {
        val_type key = dist(gen);
        // Check if the key is already present
        if (keys.find(key) == keys.end()) {
           val_type val=static_cast<val_type>(key/2);
            src.push_back({key,val});
            keys.insert(key);
        }
    }
}

auto generateNonDuplicatePairs(key_vec &keys,val_vec& vals,const size_t size)->void {
    std::unordered_set<int> unique_keys;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<val_type> dist(1, std::numeric_limits<val_type>::max());
    keys.clear();
    vals.clear();
    while (keys.size() < size) {
        val_type key = dist(gen);
        // Check if the key is already present
        if (unique_keys.find(key) == unique_keys.end()) {
           val_type val=static_cast<val_type>(key/2);
            keys.push_back(key);
            vals.push_back(val);
            unique_keys.insert(key);
        }
    }
}

template <class Fn, class ... Args>
auto timeMe(Fn fn, Args && ... args){
   std::chrono::time_point<std::chrono::_V2::system_clock, std::chrono::_V2::system_clock::duration> start,stop;
   double total_time=0;
   start = std::chrono::high_resolution_clock::now();
   fn(args...);
   stop = std::chrono::high_resolution_clock::now();
   auto duration = duration_cast<milliseconds>(stop- start).count();
   total_time+=duration;
   return total_time;
}

void benchInsert(hashmap& hmap,key_type* gpuKeys, val_type* gpuVals,int sz,float deleteRatio){
   hmap.insert(gpuKeys,gpuVals,1<<sz,1);
   hmap.erase(gpuKeys,0.5*( 1<<sz ));
   hmap.insert(gpuKeys,gpuVals,1<<sz,1);
   //hmap.insert(gpuKeys,gpuVals,1<<sz,1);
   hmap.stats();
   //hmap.erase(gpuKeys,deleteRatio*(1<<sz));
   //hmap.insert(gpuKeys,gpuVals,1<<sz);
   return ;
}

void benchInsert2(hashmap& hmap,vector& src,key_vec& keys, val_vec& vals,int sz,float deleteRatio){
   hmap.insert(src.data(),1<<sz,1);
   hmap.stats();
   hmap.erase(keys.data(),deleteRatio*(1<<sz));
   hmap.stats();
   hmap.insert(src.data(),1<<sz,1);
   hmap.stats();
   return ;
}

int main(int argc, char* argv[]){

   int sz= 14;
   float deleteRatio = 0.5;
   if (argc>=2){
      deleteRatio = atof(argv[1]);
   }
   hashmap hmap(sz+1);
   hmap.optimizeGPU();
   vector cpu_src;
   key_vec cpu_keys;
   val_vec cpu_vals;
   generateNonDuplicatePairs(cpu_keys,cpu_vals,1<<sz);
   generateNonDuplicatePairs(cpu_src,1<<sz);
   cpu_src.optimizeGPU();
   std::cout<<"Generated "<<cpu_keys.size()<<" unique keys!"<<std::endl;

   key_type* gpuKeys;
   val_type* gpuVals;
   split_gpuMalloc((void **) &gpuKeys, (1<<sz)*sizeof(key_type));
   split_gpuMalloc((void **) &gpuVals, (1<<sz)*sizeof(val_type));
   split_gpuMemcpy(gpuKeys,cpu_keys.data(),(1<<sz)*sizeof(key_type),split_gpuMemcpyHostToDevice);
   split_gpuMemcpy(gpuVals,cpu_vals.data(),(1<<sz)*sizeof(key_type),split_gpuMemcpyHostToDevice);

   key_vec keyBuffer;
   val_vec valBuffer;
   for (auto& i : cpu_src){
      keyBuffer.push_back(i.first);
      valBuffer.push_back(i.first);
   }
   keyBuffer.optimizeGPU();
   valBuffer.optimizeGPU();

   double t={0};
   for (int i =0; i<R; i++){
      hmap.optimizeGPU();
      keyBuffer.optimizeGPU();
      valBuffer.optimizeGPU();
      cpu_src.optimizeGPU();
      //t+=timeMe(benchInsert,hmap,gpuKeys,gpuVals,sz,deleteRatio);
       t+= timeMe(benchInsert2,hmap,cpu_src,keyBuffer,valBuffer,sz,deleteRatio);
      hmap.clear();
   }
   std::cout<<"Done in "<<t/R<<" ms"<<std::endl;

   split_gpuFree(gpuKeys);
   split_gpuFree(gpuVals);
   return 0;

}
