#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <chrono>
#include <limits>
#include <random>
#include "../../include/splitvector/splitvec.h"
#include "../../include/splitvector/split_tools.h"
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <nvToolsExt.h>
using namespace std::chrono;
using type_t = uint32_t;
using  splitvector = split::SplitVector<type_t> ;
using  thrustvector = thrust::device_vector<type_t> ;
constexpr int R = 100;
#define PROFILE_START(msg)   nvtxRangePushA((msg))
#define PROFILE_END() nvtxRangePop()

template <class Fn, class ... Args>
auto timeMe(Fn fn, Args && ... args){
   std::chrono::time_point<std::chrono::_V2::system_clock, std::chrono::_V2::system_clock::duration> start,stop;
   double total_time=0;
   start = std::chrono::high_resolution_clock::now();
   fn(args...);
   stop = std::chrono::high_resolution_clock::now();
   auto duration = duration_cast<microseconds>(stop- start).count();
   total_time+=duration;
   return total_time;
}

template <typename T>
void fillVec(T& vec,size_t sz){
   std::random_device dev;
   std::mt19937 rng(dev());
   std::uniform_int_distribution<std::mt19937::result_type> dist(0,std::numeric_limits<type_t>::max());
   for (size_t i=0; i< sz;++i){
      vec[i]=i;//dist(rng);
   }
   return;
}


void stream_compaction_split_old(splitvector& v,splitvector& output,  type_t* stack, size_t sz){
   auto pred =[]__host__ __device__ (type_t  element)->bool{ return (element%2)==0 ;};
   auto len = split::tools::copy_if_small(v.data(),output.data(),sz,pred,(void*)stack,sz);
}



void stream_compaction_split(splitvector& v,splitvector& output,  type_t* stack, size_t sz){
   auto pred =[]__host__ __device__ (type_t  element)->bool{ return (element%2)==0 ;};
   auto len = split::tools::copy_if_small2(v.data(),output.data(),sz,pred,(void*)stack,sz);
}
void stream_compaction_thrust(thrustvector& v,thrustvector& output){
   auto pred =[]__host__ __device__ (type_t  element)->bool{ return (element%2)==0 ;};
   thrust::copy_if(thrust::device, v.begin(), v.end(), output.begin(), pred);
}

int main(int argc, char* argv[]){



   int sz=6;
   if (argc>=2){
      sz=atoi(argv[1]);
   }
   size_t N = 64;
   splitvector v0(N),v0_out(N);
   srand(1);
   fillVec(v0,N);
   splitvector stack(N);

   v0.optimizeGPU();
   v0_out.optimizeGPU();
   stack.optimizeGPU();
   split_gpuDeviceSynchronize();
   
   double t_split={0};
   double t_split_old={0};

   for (size_t i =0 ; i < R ; ++i){
      PROFILE_START("SPLIT");
      t_split+=timeMe(stream_compaction_split,v0,v0_out,stack.data(),N);
      PROFILE_END();
      splitvector vv(v0_out);

      PROFILE_START("SPLITOLD");
      t_split_old+=timeMe(stream_compaction_split_old,v0,v0_out,stack.data(),N);
      PROFILE_END();
      assert(vv==v0_out);
   }

   printf("%d\t%f,%f\n",sz,t_split_old/R,t_split/R);

   return 0;
}









